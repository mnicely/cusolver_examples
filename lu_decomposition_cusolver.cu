#include <cassert>
#include <cstdio>
#include <stdexcept>
#include <string>

#include <hiprand.h>
#include <hipsolver.h>

#include "utilities.h"

#define VERIFY 0

constexpr int pivot_on { 1 };

template<typename T, typename U>
void SingleGPUManaged( const int &device, const U &N, const U &lda, const U &ldb, T *A, T *B ) {

    std::printf( "\ncuSolver: SingleGPUManaged GETRF\n" );

    size_t sizeBytesA { sizeof( T ) * lda * N };
    size_t sizeBytesB { sizeof( T ) * N };

#if VERIFY
    T *B_input {};
    T *A_input {};

    CUDA_RT_CALL( hipMallocManaged( &A_input, sizeBytesA ) );
    CUDA_RT_CALL( hipMallocManaged( &B_input, sizeBytesB ) );

    CUDA_RT_CALL( hipMemPrefetchAsync( A_input, sizeBytesA, hipCpuDeviceId, NULL ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( B_input, sizeBytesB, hipCpuDeviceId, NULL ) );

    CUDA_RT_CALL( hipMemcpy( A_input, A, sizeBytesA, hipMemcpyDeviceToHost ) );
    CUDA_RT_CALL( hipMemcpy( B_input, B, sizeBytesB, hipMemcpyDeviceToHost ) );
#endif

    // Start timer
    hipEvent_t startEvent { nullptr };
    hipEvent_t stopEvent { nullptr };
    float       elapsed_gpu_ms {};

    CUDA_RT_CALL( hipEventCreate( &startEvent, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stopEvent, hipEventBlockingSync ) );

    if ( pivot_on ) {
        std::printf( "Pivot is on : compute P*A = L*U\n" );
    } else {
        std::printf( "Pivot is off: compute A = L*U (not numerically stable)\n" );
    }

    /* step 1: create cusolver handle, bind a stream */
    hipsolverHandle_t cusolverH { nullptr };
    CUDA_RT_CALL( hipsolverDnCreate( &cusolverH ) );

    // Create stream
    hipStream_t stream {};
    CUDA_RT_CALL( hipStreamCreate( &stream ) );
    CUDA_RT_CALL( hipsolverSetStream( cusolverH, stream ) );

    /* step 2: copy A to device */
    int *d_info { nullptr }; /* error info */
    CUDA_RT_CALL( hipMallocManaged( &d_info, sizeof( int ) ) );

    U *d_Ipiv { nullptr }; /* pivoting sequence */
    if ( pivot_on ) {
        CUDA_RT_CALL( hipMallocManaged( &d_Ipiv, sizeof( U ) * N ) );
        CUDA_RT_CALL( hipMemPrefetchAsync( d_Ipiv, sizeof( U ) * N, device, stream ) );
    }

    void *bufferOnDevice { nullptr };
    void *bufferOnHost { nullptr };

    size_t workspaceInBytesOnDevice {};
    size_t workspaceInBytesOnHost {};

    // CUDA_RT_CALL( hipsolverDnDgetrf_bufferSize( cusolverH, N, N, A, lda, &lwork ) );
    CUDA_RT_CALL( hipsolverDnXgetrf_bufferSize(
        cusolverH, NULL, N, N, HIP_R_64F, A, lda, HIP_R_64F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost ) );

    CheckMemoryUsed( 1 );

    std::printf( "\nAllocate device workspace, lwork = %lu\n", workspaceInBytesOnDevice );
    std::printf( "Allocate host workspace, lwork = %lu\n\n", workspaceInBytesOnHost );

    CUDA_RT_CALL( hipMallocManaged( &bufferOnDevice, workspaceInBytesOnDevice ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( bufferOnDevice, workspaceInBytesOnDevice, device, stream ) );
    CUDA_RT_CALL(
        hipMemAdvise( bufferOnDevice, workspaceInBytesOnDevice, hipMemAdviseSetPreferredLocation, device ) );
    CUDA_RT_CALL( hipMemAdvise( bufferOnDevice, workspaceInBytesOnDevice, hipMemAdviseSetAccessedBy, device ) );

    if ( 0 < workspaceInBytesOnHost ) {
        CUDA_RT_CALL( hipMallocManaged( &bufferOnHost, workspaceInBytesOnHost ) );
        CUDA_RT_CALL( hipMemPrefetchAsync( bufferOnHost, workspaceInBytesOnHost, hipCpuDeviceId, NULL ) );
        assert( NULL != bufferOnHost );
    }

    // Check GPU memory used on single GPU
    CheckMemoryUsed( 1 );

    CUDA_RT_CALL( hipMemAdvise( A, sizeBytesA, hipMemAdviseSetPreferredLocation, device ) );
    CUDA_RT_CALL( hipMemAdvise( A, sizeBytesA, hipMemAdviseSetAccessedBy, device ) );

    CUDA_RT_CALL( hipMemAdvise( B, sizeBytesB, hipMemAdviseSetPreferredLocation, device ) );
    CUDA_RT_CALL( hipMemAdvise( B, sizeBytesB, hipMemAdviseSetAccessedBy, device ) );

    CUDA_RT_CALL( hipEventRecord( startEvent ) );

    /* step 4: LU factorization */
    if ( pivot_on ) {
        CUDA_RT_CALL( hipsolverDnXgetrf( cusolverH,
                                        NULL,
                                        N,
                                        N,
                                        HIP_R_64F,
                                        A,
                                        lda,
                                        d_Ipiv,
                                        HIP_R_64F,
                                        bufferOnDevice,
                                        workspaceInBytesOnDevice,
                                        bufferOnHost,
                                        workspaceInBytesOnHost,
                                        d_info ) );
    } else {
        CUDA_RT_CALL( hipsolverDnXgetrf( cusolverH,
                                        NULL,
                                        N,
                                        N,
                                        HIP_R_64F,
                                        A,
                                        lda,
                                        nullptr,
                                        HIP_R_64F,
                                        bufferOnDevice,
                                        workspaceInBytesOnDevice,
                                        bufferOnHost,
                                        workspaceInBytesOnHost,
                                        d_info ) );
    }

    // Must be here to retrieve d_info
    CUDA_RT_CALL( hipStreamSynchronize( stream ) );

    if ( *d_info ) {
        throw std::runtime_error( std::to_string( -*d_info ) + "-th parameter is wrong (hipsolverDnDgetrf) \n" );
    }

    CUDA_RT_CALL( hipMemAdvise( A, sizeBytesA, hipMemAdviseSetReadMostly, device ) );
    CUDA_RT_CALL( hipMemAdvise( d_Ipiv, sizeof( U ) * N, hipMemAdviseSetReadMostly, device ) );

    /*
     * step 5: solve A*X = B
     */

    if ( pivot_on ) {
        CUDA_RT_CALL( hipsolverDnXgetrs( cusolverH,
                                        NULL,
                                        HIPBLAS_OP_N,
                                        N,
                                        1, /* nrhs */
                                        HIP_R_64F,
                                        A,
                                        lda,
                                        d_Ipiv,
                                        HIP_R_64F,
                                        B,
                                        ldb,
                                        d_info ) );
    } else {
        CUDA_RT_CALL( hipsolverDnXgetrs( cusolverH,
                                        NULL,
                                        HIPBLAS_OP_N,
                                        N,
                                        1, /* nrhs */
                                        HIP_R_64F,
                                        A,
                                        lda,
                                        nullptr,
                                        HIP_R_64F,
                                        B,
                                        ldb,
                                        d_info ) );
    }

    // Must be here to retrieve d_info
    CUDA_RT_CALL( hipStreamSynchronize( stream ) );

    if ( *d_info ) {
        throw std::runtime_error( std::to_string( -*d_info ) + "-th parameter is wrong (hipsolverDnDgetrs) \n" );
    }

    // Stop timer
    CUDA_RT_CALL( hipEventRecord( stopEvent ) );
    CUDA_RT_CALL( hipEventSynchronize( stopEvent ) );

    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent ) );
    std::printf( "\nRuntime = %0.2f ms\n\n", elapsed_gpu_ms );

#if VERIFY
    CUDA_RT_CALL( hipMemPrefetchAsync( B, sizeBytesB, hipCpuDeviceId, stream ) );

    // Calculate Residual Error
    CalculateResidualError( N, lda, A_input, B_input, B );
#endif

    if ( d_Ipiv )
        CUDA_RT_CALL( hipFree( d_Ipiv ) );
    if ( d_info )
        CUDA_RT_CALL( hipFree( d_info ) );
    if ( bufferOnDevice )
        CUDA_RT_CALL( hipFree( bufferOnDevice ) );
    if ( bufferOnHost )
        CUDA_RT_CALL( hipFree( bufferOnHost ) );
    if ( cusolverH )
        CUDA_RT_CALL( hipsolverDnDestroy( cusolverH ) );
    if ( stream )
        CUDA_RT_CALL( hipStreamDestroy( stream ) );
}

int main( int argc, char *argv[] ) {

    int device = -1;
    CUDA_RT_CALL( hipGetDevice( &device ) );

    const int64_t m { 39000 };
    const int64_t lda { m };
    const int64_t ldb { m };

    double *m_A {};
    double *m_B {};

    CUDA_RT_CALL( hipMallocManaged( &m_A, sizeof( double ) * lda * m ) );
    CUDA_RT_CALL( hipMallocManaged( &m_B, sizeof( double ) * m ) );

    CUDA_RT_CALL( hipMemPrefetchAsync( m_A, sizeof( double ) * lda * m, device, NULL ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( m_B, sizeof( double ) * m, device, NULL ) );

    // Generate random numbers on the GPU
    hiprandGenerator_t gen;
    CUDA_RT_CALL( hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_DEFAULT ) );
    CUDA_RT_CALL( hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL ) );
    std::printf(
        "Number generation of %lu values (A): %lu\n", static_cast<size_t>( m ) * lda, sizeof( double ) * lda * m );
    CUDA_RT_CALL( hiprandGenerateNormalDouble( gen, m_A, static_cast<size_t>( m ) * lda, 100.0, 50.0 ) );
    std::printf( "Number generation of %lu values (B): %lu\n", static_cast<size_t>( m ), sizeof( double ) * m );
    CUDA_RT_CALL( hiprandGenerateNormalDouble( gen, m_B, static_cast<size_t>( m ), 100.0, 50.0 ) );

    // Managed Memory
    std::printf( "Run LU Decomposition\n" );
    SingleGPUManaged( device, m, lda, ldb, m_A, m_B );

    CUDA_RT_CALL( hipFree( m_A ) );
    CUDA_RT_CALL( hipFree( m_B ) );

    return ( EXIT_SUCCESS );
}