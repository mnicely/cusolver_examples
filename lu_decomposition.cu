#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <stdexcept>
#include <vector>

#include <hiprand.h>
#include <hipsolver.h>
#include <hipsolver.h>

#include "utilities.h"

#define VERIFY 0

constexpr int pivot_on { 1 };

void SingleGPUManaged( const size_t &N, const size_t &lda, const size_t &ldb, double *A, double *B, double *X ) {

    std::printf( "\ncuSolver: SingleGPUManaged GETRF\n" );

#if VERIFY
    double *B_input {};
    double *A_input {};

    CUDA_RT_CALL( hipMallocManaged( &A_input, sizeof( double ) * lda * N ) );
    CUDA_RT_CALL( hipMallocManaged( &B_input, sizeof( double ) * N ) );

    for ( int i = 0; i < N; i++ ) {
        B_input[i] = B[i];
    }

    for ( int i = 0; i < lda * N; i++ ) {
        A_input[i] = A[i];
    }
#endif

    // Start timer
    hipEvent_t startEvent { nullptr };
    hipEvent_t stopEvent { nullptr };
    float       elapsed_gpu_ms {};

    CUDA_RT_CALL( hipEventCreate( &startEvent, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stopEvent, hipEventBlockingSync ) );

    int lwork {}; /* size of workspace */

    int *d_Ipiv { nullptr }; /* pivoting sequence */
    int *d_info { nullptr }; /* error info */

    double *d_work { nullptr }; /* device workspace for getrf */

    if ( pivot_on ) {
        std::printf( "Pivot is on : compute P*A = L*U\n" );
    } else {
        std::printf( "Pivot is off: compute A = L*U (not numerically stable)\n" );
    }

    /* step 1: create cusolver handle, bind a stream */
    hipsolverHandle_t cusolverH { nullptr };
    CUDA_RT_CALL( hipsolverDnCreate( &cusolverH ) );

    // Create stream
    hipStream_t stream {};
    CUDA_RT_CALL( hipStreamCreate( &stream ) );
    CUDA_RT_CALL( hipsolverSetStream( cusolverH, stream ) );

    /* step 2: copy A to device */
    CUDA_RT_CALL( hipMallocManaged( &d_info, sizeof( int ) ) );

    if ( pivot_on ) {
        CUDA_RT_CALL( hipMallocManaged( &d_Ipiv, sizeof( int ) * N ) );
        CUDA_RT_CALL( hipMemPrefetchAsync( d_Ipiv, N * sizeof( int ), 0, stream ) );
    }

    CUDA_RT_CALL( hipsolverDnDgetrf_bufferSize( cusolverH, N, N, A, lda, &lwork ) );

    CheckMemoryUsed( 1 );

    std::printf( "lwork = %d\n", lwork );
    std::printf( "\tAllocate device workspace, lwork = %lu\n", sizeof( double ) * lwork );

    CUDA_RT_CALL( hipMallocManaged( &d_work, ( sizeof( double ) * lwork ) ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( d_work, sizeof( double ) * lwork, 0, stream ) );

    // Check GPU memory used on single GPU
    CheckMemoryUsed( 1 );

    CUDA_RT_CALL( hipEventRecord( startEvent ) );

    /* step 4: LU factorization */
    if ( pivot_on ) {
        CUDA_RT_CALL( hipsolverDnDgetrf( cusolverH, N, N, A, lda, d_work, d_Ipiv, d_info ) );
    } else {
        CUDA_RT_CALL( hipsolverDnDgetrf( cusolverH, N, N, A, lda, d_work, nullptr, d_info ) );
    }

    // Must be here to retrieve d_info
    CUDA_RT_CALL( hipStreamSynchronize( stream ) );

    if ( *d_info ) {
        throw std::runtime_error( std::to_string( -*d_info ) + "-th parameter is wrong (hipsolverDnDgetrf) \n" );
    }

    /*
     * step 5: solve A*X = B
     */
    if ( pivot_on ) {
        CUDA_RT_CALL( hipsolverDnDgetrs( cusolverH,
                                        HIPBLAS_OP_N,
                                        N,
                                        1, /* nrhs */
                                        A,
                                        lda,
                                        d_Ipiv,
                                        B,
                                        ldb,
                                        d_info ) );
    } else {
        CUDA_RT_CALL( hipsolverDnDgetrs( cusolverH,
                                        HIPBLAS_OP_N,
                                        N,
                                        1, /* nrhs */
                                        A,
                                        lda,
                                        nullptr,
                                        B,
                                        ldb,
                                        d_info ) );
    }

    // Must be here to retrieve d_info
    CUDA_RT_CALL( hipStreamSynchronize( stream ) );

    if ( *d_info ) {
        throw std::runtime_error( std::to_string( -*d_info ) + "-th parameter is wrong (hipsolverDnDgetrs) \n" );
    }

    // Stop timer
    CUDA_RT_CALL( hipEventRecord( stopEvent ) );
    CUDA_RT_CALL( hipEventSynchronize( stopEvent ) );

    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent ) );
    std::printf( "\nRuntime = %0.2f ms\n\n", elapsed_gpu_ms );

    if ( d_Ipiv )
        CUDA_RT_CALL( hipFree( d_Ipiv ) );
    if ( d_info )
        CUDA_RT_CALL( hipFree( d_info ) );
    if ( d_work )
        CUDA_RT_CALL( hipFree( d_work ) );
    if ( cusolverH )
        CUDA_RT_CALL( hipsolverDnDestroy( cusolverH ) );

    CUDA_RT_CALL( hipMemPrefetchAsync( B, sizeof( double ) * N, hipCpuDeviceId, stream ) );

#if VERIFY
    for ( int i = 0; i < N; i++ ) {
        X[i] = B[i];
    }

    // Calculate Residual Error
    CalculateResidualError( N, lda, A_input, B_input, X );
#endif
}

int main( int argc, char *argv[] ) {

    int device = -1;
    CUDA_RT_CALL( hipGetDevice( &device ) );

    const size_t m { 39000 };
    const size_t lda { m };
    const size_t ldb { m };

    double *m_A {};
    double *m_B {};
    double *m_single_X {};

    CUDA_RT_CALL( hipMallocManaged( &m_A, sizeof( double ) * lda * m ) );
    CUDA_RT_CALL( hipMallocManaged( &m_B, sizeof( double ) * m ) );
    CUDA_RT_CALL( hipMallocManaged( &m_single_X, sizeof( double ) * m ) );

    CUDA_RT_CALL( hipMemPrefetchAsync( m_A, sizeof( double ) * lda * m, device, NULL ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( m_B, sizeof( double ) * m, device, NULL ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( m_single_X, sizeof( double ) * m, hipCpuDeviceId, NULL ) );
    // CUDA_RT_CALL( hipMemPrefetchAsync( m_single_X, sizeof( double ) * m, device, NULL ) );

    // Generate random numbers on the GPU
    hiprandGenerator_t gen;
    CUDA_RT_CALL( hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_DEFAULT ) );
    CUDA_RT_CALL( hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL ) );
    std::printf( "Number generation of %lu values (A): %lu\n", lda * m, sizeof( double ) * lda * m );
    CUDA_RT_CALL( hiprandGenerateNormalDouble( gen, m_A, lda * m, 100.0, 50.0 ) );
    std::printf( "Number generation of %lu values (B): %lu\n", m, sizeof( double ) * m );
    CUDA_RT_CALL( hiprandGenerateNormalDouble( gen, m_B, m, 100.0, 50.0 ) );

    // Managed Memory
    std::printf( "Run LU Decomposition\n" );
    SingleGPUManaged( m, lda, ldb, m_A, m_B, m_single_X );

    CUDA_RT_CALL( hipFree( m_A ) );
    CUDA_RT_CALL( hipFree( m_B ) );
    CUDA_RT_CALL( hipFree( m_single_X ) );

    return ( EXIT_SUCCESS );
}