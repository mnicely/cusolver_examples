// Redistribution and use in source and binary forms, with or without modification, are permitted
// provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright notice, this list of
//       conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright notice, this list of
//       conditions and the following disclaimer in the documentation and/or other materials
//       provided with the distribution.
//     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
//       to endorse or promote products derived from this software without specific prior written
//       permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
// IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
// FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
// OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
// STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <cassert>
#include <cstdio>
#include <cstring>
#include <stdexcept>

#include <hiprand.h>

#include "magma_v2.h"

#include "utilities.h"

#define VERIFY 0

template<typename T, typename U>
void SingleGPUManaged( const int &ngpu, const int &loops, const U &N, const U &lda, const U &ldb, T *A, T *B ) {

#if VERIFY
    T *B_input {};
    T *A_input {};

    size_t sizeBytesA { sizeof( T ) * lda * N };
    size_t sizeBytesB { sizeof( T ) * N };

    CUDA_RT_CALL( hipMallocManaged( &A_input, sizeBytesA ) );
    CUDA_RT_CALL( hipMallocManaged( &B_input, sizeBytesB ) );

    CUDA_RT_CALL( hipMemPrefetchAsync( A_input, sizeBytesA, hipCpuDeviceId, NULL ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( B_input, sizeBytesB, hipCpuDeviceId, NULL ) );

    CUDA_RT_CALL( hipMemcpy( A_input, A, sizeBytesA, hipMemcpyDeviceToHost ) );
    CUDA_RT_CALL( hipMemcpy( B_input, B, sizeBytesB, hipMemcpyDeviceToHost ) );
#endif

    // Start timer
    hipEvent_t startEvent { nullptr };
    hipEvent_t stopEvent { nullptr };
    float       elapsed_gpu_ms {};

    CUDA_RT_CALL( hipEventCreate( &startEvent, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stopEvent, hipEventBlockingSync ) );

    std::printf( "Pivot is on : compute P*A = L*U\n" );

    magma_int_t info {}; /* error info */

    U *d_Ipiv { nullptr }; /* pivoting sequence */
    if ( MAGMA_SUCCESS != magma_imalloc_cpu( &d_Ipiv, N ) ) {
        throw std::runtime_error( "Error allocating d_Ipiv\n" );
    }

    // Check GPU memory used on single GPU
    CheckMemoryUsed( 1 );

    std::printf( "\nRunning GETRF\n" );

    CUDA_RT_CALL( hipEventRecord( startEvent ) );

    for ( int i = 0; i < loops; i++ ) {

        /* step 4: LU factorization */
        if ( ngpu > 1 ) {
            CUDA_RT_CALL( magma_zgetrf_m( ngpu, N, N, A, lda, d_Ipiv, &info ) );
        } else {
            CUDA_RT_CALL( magma_zgetrf( N, N, A, lda, d_Ipiv, &info ) );
        }

        if ( info != 0 ) {
            throw std::runtime_error( std::to_string( -info ) + "-th parameter is wrong (magma_zgetrf) \n" );
        }
    }

    CUDA_RT_CALL( hipDeviceSynchronize( ) );

    // Stop timer
    CUDA_RT_CALL( hipEventRecord( stopEvent ) );
    CUDA_RT_CALL( hipEventSynchronize( stopEvent ) );

    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent ) );
    double avg { elapsed_gpu_ms / loops };
    double flops { FLOPS_ZGETRF( N, N ) };
    double perf { 1e-9 * flops / avg };
    std::printf( "\nRuntime = %0.2f ms (avg over %d runs) : @ %0.2f GFLOPs\n\n", avg, loops, perf );

#if VERIFY
    CUDA_RT_CALL( hipMemPrefetchAsync( B, sizeBytesB, hipCpuDeviceId, stream ) );

    // Calculate Residual Error
    CalculateResidualError( N,
                            lda,
                            reinterpret_cast<double *>( A_input ),
                            reinterpret_cast<double *>( B_input ),
                            reinterpret_cast<double *>( B ) );
#endif

    if ( MAGMA_SUCCESS != magma_free_cpu( d_Ipiv ) ) {
        throw std::runtime_error( "Error freeing d_Ipiv\n" );
    }
    CUDA_RT_CALL( hipEventDestroy( startEvent ) );
    CUDA_RT_CALL( hipEventDestroy( stopEvent ) );
#if VERIFY
    CUDA_RT_CALL( hipFree( A_input ) );
    CUDA_RT_CALL( hipFree( B_input ) );
#endif
}

int main( int argc, char *argv[] ) {

    if ( MAGMA_SUCCESS != magma_init( ) ) {
        throw std::runtime_error( "Error magma_init\n" );
    }

    magma_int_t m {};
    magma_int_t loops {};
    if ( argc < 3 ) {
        m     = 512;
        loops = 5;
    } else {
        m     = std::atoi( argv[1] );
        loops = std::atoi( argv[2] );
    }

    magma_int_t ngpu = magma_num_gpus( );
    std::printf( "Magma sees %llu GPUs\n", ngpu );

    const magma_int_t lda { m };
    const magma_int_t ldb { m };

    using data_type = magmaDoubleComplex;

    data_type *temp_A {};
    data_type *temp_B {};

    size_t sizeA { static_cast<size_t>( lda ) * m };
    size_t sizeB { static_cast<size_t>( m ) };

    CUDA_RT_CALL( hipMallocManaged( &temp_A, sizeof( data_type ) * sizeA ) );
    CUDA_RT_CALL( hipMallocManaged( &temp_B, sizeof( data_type ) * sizeB ) );

    // Generate random numbers on the GPU
    CreateRandomData( "A", sizeA * 2, reinterpret_cast<double *>( temp_A ) );
    CreateRandomData( "B", sizeB * 2, reinterpret_cast<double *>( temp_B ) );

    data_type *m_A {};
    data_type *m_B {};

    if ( MAGMA_SUCCESS != magma_zmalloc_pinned( &m_A, sizeA ) ) {
        throw std::runtime_error( "Error allocating A\n" );
    }
    if ( MAGMA_SUCCESS != magma_zmalloc_pinned( &m_B, sizeB ) ) {
        throw std::runtime_error( "Error allocating B\n" );
    }

    std::memcpy( m_A, temp_A, sizeof( data_type ) * sizeA );
    std::memcpy( m_B, temp_B, sizeof( data_type ) * sizeB );

    // Free memory
    CUDA_RT_CALL( hipFree( temp_A ) );
    CUDA_RT_CALL( hipFree( temp_B ) );

    // Managed Memory
    for ( int i = 1; i < ( ngpu * 2 ); i *= 2 ) {
        std::printf( "\n\n******************************************\n" );
        std::printf( "Run Warmup w/ %d GPUs\n", i );
        SingleGPUManaged( ngpu, 1, m, lda, ldb, m_A, m_B );

        std::printf( "\n\n******************************************\n" );
        std::printf( "Run LU Decomposition w/ %d GPUs\n", i );
        SingleGPUManaged( ngpu, loops, m, lda, ldb, m_A, m_B );
    }

    if ( MAGMA_SUCCESS != magma_free_pinned( m_A ) ) {
        throw std::runtime_error( "Error freeing A\n" );
    }

    if ( MAGMA_SUCCESS != magma_free_pinned( m_B ) ) {
        throw std::runtime_error( "Error freeing B\n" );
    }

    return ( EXIT_SUCCESS );
}