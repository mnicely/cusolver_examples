#include <cassert>
#include <cstdio>
#include <stdexcept>
#include <string>

#include <hiprand.h>

#include "magma_v2.h"
// #include "magma_lapack.h"

#include "utilities.h"

#define VERIFY 0

constexpr int pivot_on { 1 };

template<typename T, typename U>
void SingleGPUManaged( const int &device, const U &N, const U &lda, const U &ldb, T *A, T *B ) {

    std::printf( "\ncuSolver: SingleGPUManaged GETRF\n" );

#if VERIFY
	size_t sizeBytesA { sizeof( T ) * lda * N };
    size_t sizeBytesB { sizeof( T ) * N };

    T *B_input {};
    T *A_input {};

    CUDA_RT_CALL( hipMallocManaged( &A_input, sizeBytesA ) );
    CUDA_RT_CALL( hipMallocManaged( &B_input, sizeBytesB ) );

    CUDA_RT_CALL( hipMemPrefetchAsync( A_input, sizeBytesA, hipCpuDeviceId, NULL ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( B_input, sizeBytesB, hipCpuDeviceId, NULL ) );

    CUDA_RT_CALL( hipMemcpy( A_input, A, sizeBytesA, hipMemcpyDeviceToHost ) );
    CUDA_RT_CALL( hipMemcpy( B_input, B, sizeBytesB, hipMemcpyDeviceToHost ) );
#endif

    // Start timer
    hipEvent_t startEvent { nullptr };
    hipEvent_t stopEvent { nullptr };
    float       elapsed_gpu_ms {};

    CUDA_RT_CALL( hipEventCreate( &startEvent, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stopEvent, hipEventBlockingSync ) );

    if ( pivot_on ) {
        std::printf( "Pivot is on : compute P*A = L*U\n" );
    } else {
        std::printf( "Pivot is off: compute A = L*U (not numerically stable)\n" );
    }

	magma_init();

    /* step 1: create cusolver handle, bind a stream */

//     // Create stream
    hipStream_t stream {};
    CUDA_RT_CALL( hipStreamCreate( &stream ) );

    /* step 2: copy A to device */
    int *d_info { nullptr }; /* error info */
    CUDA_RT_CALL( hipMallocManaged( &d_info, sizeof( int ) ) );

    U *d_Ipiv { nullptr }; /* pivoting sequence */
    if ( pivot_on ) {
        CUDA_RT_CALL( hipMallocManaged( &d_Ipiv, sizeof( U ) * N ) );
        CUDA_RT_CALL( hipMemPrefetchAsync( d_Ipiv, sizeof( U ) * N, device, NULL ) );
    }

    CheckMemoryUsed( 1 );

    // Check GPU memory used on single GPU
    CheckMemoryUsed( 1 );

    CUDA_RT_CALL( hipEventRecord( startEvent ) );

    /* step 4: LU factorization */
    if ( pivot_on ) {
        CUDA_RT_CALL( magma_dgetrf_gpu( N, N, A, lda, d_Ipiv, d_info ) );
    } else {
        CUDA_RT_CALL( magma_dgetrf_nopiv_gpu( N, N, A, lda, d_info ) );
    }

    // Must be here to retrieve d_info
    CUDA_RT_CALL( hipStreamSynchronize( stream ) );

    if ( *d_info ) {
        throw std::runtime_error( std::to_string( -*d_info ) + "-th parameter is wrong (hipsolverDnDgetrf) \n" );
    }

    /*
     * step 5: solve A*X = B
     */

    if ( pivot_on ) {
        CUDA_RT_CALL( magma_dgetrs_gpu(
			MagmaNoTrans,
                                        N,
                                        1, /* nrhs */
                                        A,
                                        lda,
                                        d_Ipiv,
                                        B,
                                        ldb,
                                        d_info ) );
    } else {
        CUDA_RT_CALL( magma_dgetrs_nopiv_gpu( 
			MagmaNoTrans,
                                        N,
                                        1, /* nrhs */
                                        A,
                                        lda,
                                        B,
                                        ldb,
                                        d_info ) );
    }

    // Must be here to retrieve d_info
    CUDA_RT_CALL( hipStreamSynchronize( stream ) );

    if ( *d_info ) {
        throw std::runtime_error( std::to_string( -*d_info ) + "-th parameter is wrong (hipsolverDnDgetrs) \n" );
    }

    // Stop timer
    CUDA_RT_CALL( hipEventRecord( stopEvent ) );
    CUDA_RT_CALL( hipEventSynchronize( stopEvent ) );

    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent ) );
    std::printf( "\nRuntime = %0.2f ms\n\n", elapsed_gpu_ms );

#if VERIFY
    CUDA_RT_CALL( hipMemPrefetchAsync( B, sizeBytesB, hipCpuDeviceId, stream ) );

    // Calculate Residual Error
    CalculateResidualError( N, lda, A_input, B_input, B );
#endif

    if ( d_Ipiv )
        CUDA_RT_CALL( hipFree( d_Ipiv ) );
    if ( d_info )
        CUDA_RT_CALL( hipFree( d_info ) );
    if ( stream )
        CUDA_RT_CALL( hipStreamDestroy( stream ) );
}

int main( int argc, char *argv[] ) {

    int device = -1;
    CUDA_RT_CALL( hipGetDevice( &device ) );

    const magma_int_t m { 39000 };
    const magma_int_t lda { m };
    const magma_int_t ldb { m };

    double *m_A {};
    double *m_B {};

    CUDA_RT_CALL( hipMallocManaged( &m_A, sizeof( double ) * lda * m ) );
    CUDA_RT_CALL( hipMallocManaged( &m_B, sizeof( double ) * m ) );

    CUDA_RT_CALL( hipMemPrefetchAsync( m_A, sizeof( double ) * lda * m, device, NULL ) );
    CUDA_RT_CALL( hipMemPrefetchAsync( m_B, sizeof( double ) * m, device, NULL ) );

    // Generate random numbers on the GPU
    hiprandGenerator_t gen;
    CUDA_RT_CALL( hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_DEFAULT ) );
    CUDA_RT_CALL( hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL ) );
    std::printf(
        "Number generation of %lu values (A): %lu\n", static_cast<size_t>( m ) * lda, sizeof( double ) * lda * m );
    CUDA_RT_CALL( hiprandGenerateNormalDouble( gen, m_A, static_cast<size_t>( m ) * lda, 100.0, 50.0 ) );
    std::printf( "Number generation of %lu values (B): %lu\n", static_cast<size_t>( m ), sizeof( double ) * m );
    CUDA_RT_CALL( hiprandGenerateNormalDouble( gen, m_B, static_cast<size_t>( m ), 100.0, 50.0 ) );

    // Managed Memory
    std::printf( "Run LU Decomposition\n" );
    SingleGPUManaged( device, m, lda, ldb, m_A, m_B );

    CUDA_RT_CALL( hipFree( m_A ) );
    CUDA_RT_CALL( hipFree( m_B ) );

    return ( EXIT_SUCCESS );
}